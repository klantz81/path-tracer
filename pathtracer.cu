#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <vector>
#include <iostream>

#include "src/vec_device.h"
#include "src/stack_device.h"
#include "src/common.h"
#include "src/enum.h"


__device__ __intersection_t rayIntersectsAABB(const __node& n, unsigned short index, __ray& r) {

	__intersection_t isect;
	isect.intersects = false;
	isect.tmin = 0;
	isect.tmax = 1e8;

	char side = 0;

	if (r.origin.x >= n.bounds_device[index].minx && r.origin.x <= n.bounds_device[index].maxx &&
	    r.origin.y >= n.bounds_device[index].miny && r.origin.y <= n.bounds_device[index].maxy &&
	    r.origin.z >= n.bounds_device[index].minz && r.origin.z <= n.bounds_device[index].maxz) {
		isect.intersects = true;
		isect.tmin = 0;
	}
	
	if (!isect.intersects && r.origin.x < n.bounds_device[index].minx && r.direction.x > 0) {
		double t = (n.bounds_device[index].minx - r.origin.x) / r.direction.x;
		double y = r.origin.y + t * r.direction.y;
		double z = r.origin.z + t * r.direction.z;
		if (y >= n.bounds_device[index].miny && y <= n.bounds_device[index].maxy && z >= n.bounds_device[index].minz && z <= n.bounds_device[index].maxz) {
			isect.intersects = true;
			isect.tmin = t;
			side = 1;
		}
	}
	if (!isect.intersects && r.origin.x > n.bounds_device[index].maxx && r.direction.x < 0) {
		double t = (n.bounds_device[index].maxx - r.origin.x) / r.direction.x;
		double y = r.origin.y + t * r.direction.y;
		double z = r.origin.z + t * r.direction.z;
		if (y >= n.bounds_device[index].miny && y <= n.bounds_device[index].maxy && z >= n.bounds_device[index].minz && z <= n.bounds_device[index].maxz) {
			isect.intersects = true;
			isect.tmin = t;
			side = 2;
		}
	}
	if (!isect.intersects && r.origin.y < n.bounds_device[index].miny && r.direction.y > 0) {
		double t = (n.bounds_device[index].miny - r.origin.y) / r.direction.y;
		double x = r.origin.x + t * r.direction.x;
		double z = r.origin.z + t * r.direction.z;
		if (x >= n.bounds_device[index].minx && x <= n.bounds_device[index].maxx && z >= n.bounds_device[index].minz && z <= n.bounds_device[index].maxz) {
			isect.intersects = true;
			isect.tmin = t;
			side = 3;
		}
	}
	if (!isect.intersects && r.origin.y > n.bounds_device[index].maxy && r.direction.y < 0) {
		double t = (n.bounds_device[index].maxy - r.origin.y) / r.direction.y;
		double x = r.origin.x + t * r.direction.x;
		double z = r.origin.z + t * r.direction.z;
		if (x >= n.bounds_device[index].minx && x <= n.bounds_device[index].maxx && z >= n.bounds_device[index].minz && z <= n.bounds_device[index].maxz) {
			isect.intersects = true;
			isect.tmin = t;
			side = 4;
		}
	}
	if (!isect.intersects && r.origin.z < n.bounds_device[index].minz && r.direction.z > 0) {
		double t = (n.bounds_device[index].minz - r.origin.z) / r.direction.z;
		double x = r.origin.x + t * r.direction.x;
		double y = r.origin.y + t * r.direction.y;
		if (x >= n.bounds_device[index].minx && x <= n.bounds_device[index].maxx && y >= n.bounds_device[index].miny && y <= n.bounds_device[index].maxy) {
			isect.intersects = true;
			isect.tmin = t;
			side = 5;
		}
	}
	if (!isect.intersects && r.origin.z > n.bounds_device[index].maxz && r.direction.z < 0) {
		double t = (n.bounds_device[index].maxz - r.origin.z) / r.direction.z;
		double x = r.origin.x + t * r.direction.x;
		double y = r.origin.y + t * r.direction.y;
		if (x >= n.bounds_device[index].minx && x <= n.bounds_device[index].maxx && y >= n.bounds_device[index].miny && y <= n.bounds_device[index].maxy) {
			isect.intersects = true;
			isect.tmin = t;
			side = 6;
		}
	}

	if (isect.intersects) {
		double epsilon = 0.0000000000001;
		if (side != 1) {
			__vector n0(1.0,0.0,0.0);
			__vector p0(n.bounds_device[index].minx, n.bounds_device[index].miny, n.bounds_device[index].minz);
			double den = r.direction * n0;
			if (fabs(den) > epsilon) {
				double t = ((p0 - r.origin) * n0) / den;
				if (t > isect.tmin) {
					__vector _p0 = r.origin + r.direction * t;
					if (_p0.y >= n.bounds_device[index].miny && _p0.y <= n.bounds_device[index].maxy && _p0.z >= n.bounds_device[index].minz && _p0.z <= n.bounds_device[index].maxz) {
						isect.tmax = t;
					}
				}
			}
		}
		if (side != 2) {
			__vector n0(1.0,0.0,0.0);
			__vector p0(n.bounds_device[index].maxx, n.bounds_device[index].miny, n.bounds_device[index].minz);
			double den = r.direction * n0;
			if (fabs(den) > epsilon) {
				double t = ((p0 - r.origin) * n0) / den;
				if (t > isect.tmin) {
					__vector _p0 = r.origin + r.direction * t;
					if (_p0.y >= n.bounds_device[index].miny && _p0.y <= n.bounds_device[index].maxy && _p0.z >= n.bounds_device[index].minz && _p0.z <= n.bounds_device[index].maxz) {
						isect.tmax = t;
					}
				}
			}
		}
		if (side != 3) {
			__vector n0(0.0,1.0,0.0);
			__vector p0(n.bounds_device[index].minx, n.bounds_device[index].miny, n.bounds_device[index].minz);
			double den = r.direction * n0;
			if (fabs(den) > epsilon) {
				double t = ((p0 - r.origin) * n0) / den;
				if (t > isect.tmin) {
					__vector _p0 = r.origin + r.direction * t;
					if (_p0.x >= n.bounds_device[index].minx && _p0.x <= n.bounds_device[index].maxx && _p0.z >= n.bounds_device[index].minz && _p0.z <= n.bounds_device[index].maxz) {
						isect.tmax = t;
					}
				}
			}
		}
		if (side != 4) {
			__vector n0(0.0,1.0,0.0);
			__vector p0(n.bounds_device[index].minx, n.bounds_device[index].maxy, n.bounds_device[index].minz);
			double den = r.direction * n0;
			if (fabs(den) > epsilon) {
				double t = ((p0 - r.origin) * n0) / den;
				if (t > isect.tmin) {
					__vector _p0 = r.origin + r.direction * t;
					if (_p0.x >= n.bounds_device[index].minx && _p0.x <= n.bounds_device[index].maxx && _p0.z >= n.bounds_device[index].minz && _p0.z <= n.bounds_device[index].maxz) {
						isect.tmax = t;
					}
				}
			}
		}
		if (side != 5) {
			__vector n0(0.0,0.0,1.0);
			__vector p0(n.bounds_device[index].minx, n.bounds_device[index].miny, n.bounds_device[index].minz);
			double den = r.direction * n0;
			if (fabs(den) > epsilon) {
				double t = ((p0 - r.origin) * n0) / den;
				if (t > isect.tmin) {
					__vector _p0 = r.origin + r.direction * t;
					if (_p0.y >= n.bounds_device[index].miny && _p0.y <= n.bounds_device[index].maxy && _p0.x >= n.bounds_device[index].minx && _p0.x <= n.bounds_device[index].maxx) {
						isect.tmax = t;
					}
				}
			}
		}
		if (side != 6) {
			__vector n0(0.0,0.0,1.0);
			__vector p0(n.bounds_device[index].minx, n.bounds_device[index].miny, n.bounds_device[index].maxz);
			double den = r.direction * n0;
			if (fabs(den) > epsilon) {
				double t = ((p0 - r.origin) * n0) / den;
				if (t > isect.tmin) {
					__vector _p0 = r.origin + r.direction * t;
					if (_p0.y >= n.bounds_device[index].miny && _p0.y <= n.bounds_device[index].maxy && _p0.x >= n.bounds_device[index].minx && _p0.x <= n.bounds_device[index].maxx) {
						isect.tmax = t;
					}
				}
			}
		}
	}

	return isect;
}



__device__ __vector sampleRay(__node n, __ray ray, __object objects[], int object_count, int depth, float* rand_device, int rand_index, int rand_size, int max_bounces, __textures texture) {
	double epsilon = 0.0000000000001;

	__intersection intersect; // best intersection
	__intersection isect; // test intersection
	
	int which = -1;

	max_bounces = MIN(20, MAX(max_bounces, 1));
	__vector __a[20], __b[20], sample(0,0,0);

	for (int l = 0; l < max_bounces; l++) {
		__a[l].x = __a[l].y = __a[l].z = 0;
		__b[l].x = __b[l].y = __b[l].z = 0;
	}

	for (int l = 0; l < max_bounces; l++) {

		ray.direction = ray.direction.unit();
		intersect.intersects = false;

		__stack stack;
		__intersection_t root_intersection = rayIntersectsAABB(n, 0, ray);

		if (root_intersection.intersects) {
			stack.push(0, root_intersection.tmin, root_intersection.tmax);
			while (!stack.empty() && !intersect.intersects) {
				__stack_element se = stack.pop();
				while (n.leaf_id_device[se.id] < 0) {
					char axis = n.axis_device[se.id];
					double tsplit;
					int first, second;
					if (axis == Z) {
						tsplit = (n.bounds_device[n.child0_device[se.id]].maxz - ray.origin.z) / ray.direction.z; // what about rays parallel to the split plane?
						if (n.bounds_device[n.child1_device[se.id]].minz - ray.origin.z >= 0.0) {
							first = n.child0_device[se.id];
							second = n.child1_device[se.id];
						} else {
							first = n.child1_device[se.id];
							second = n.child0_device[se.id];
						}
					} else if (axis == Y) {
						tsplit = (n.bounds_device[n.child0_device[se.id]].maxy - ray.origin.y) / ray.direction.y; // what about rays parallel to the split plane?
						if (n.bounds_device[n.child1_device[se.id]].miny - ray.origin.y >= 0.0) {
							first = n.child0_device[se.id];
							second = n.child1_device[se.id];
						} else {
							first = n.child1_device[se.id];
							second = n.child0_device[se.id];
						}
					} else {
						tsplit = (n.bounds_device[n.child0_device[se.id]].maxx - ray.origin.x) / ray.direction.x; // what about rays parallel to the split plane?
						if (n.bounds_device[n.child1_device[se.id]].minx - ray.origin.x >= 0.0) {
							first = n.child0_device[se.id];
							second = n.child1_device[se.id];
						} else {
							first = n.child1_device[se.id];
							second = n.child0_device[se.id];
						}
					}

					if (tsplit >= se.tmax || tsplit < 0) {
						se.id = first;
					} else if (tsplit <= se.tmin) {
						se.id = second;
					} else {
						stack.push(second, tsplit, se.tmax);
						se.id = first;
						se.tmax = tsplit;
					}
				}

				// check intersections for se.id
				for (int j = 0; j < n.object_count_device[se.id]; j++) {
					int k = n.objects_device[n.leaf_id_device[se.id] * n.max_leaf_objects + j];

					isect.intersects = false;

					if (objects[k].type == SPHERE) {

						double a = ray.direction * ray.direction;
						double b = (ray.direction * ray.origin - ray.direction * objects[k].center) * 2.0;
						double c = ray.origin * ray.origin + objects[k].center * objects[k].center - ray.origin * objects[k].center * 2.0 - objects[k].radius * objects[k].radius;
						double det = b * b - 4 * a * c;
						if (det < epsilon) continue;

						double t0 = (-b + sqrt(det))/(2 * a);
						double t1 = (-b - sqrt(det))/(2 * a);
						if (t0 < epsilon && t1 < epsilon) continue;

						isect.intersects = true;
						isect.t = t0 < epsilon ? t1 : (t1 < epsilon ? t0 : (t0 < t1 ? t0 : t1));

						isect.ray.origin = ray.origin + ray.direction * isect.t;

						isect.normal = (isect.ray.origin - objects[k].center).unit();

					} else if (objects[k].type == TRIANGLE) {

						double den = ray.direction * objects[k].n;
						if (fabs(den) < epsilon) continue;

						double num = (objects[k].p0 - ray.origin) * objects[k].n;
						double num_den = num/den;
						if (num_den < epsilon) continue;

						__vector v0 = objects[k].p1 - objects[k].p0;
						__vector v1 = objects[k].p2 - objects[k].p0;
						__vector p = (ray.origin + ray.direction * num_den) - objects[k].p0;
							
						double pv0 = p*v0;
						double pv1 = p*v1;
						double v0v0 = v0*v0;
						double v0v1 = v0*v1;
						double v1v1 = v1*v1;
						den = v0v0*v1v1 - v0v1*v0v1;
						double s = (pv0*v1v1 - pv1*v0v1)/den;
						double t = (pv1*v0v0 - pv0*v0v1)/den;
						if (s >= 0 && t >= 0 && s+t<1.0) {

							isect.intersects = true;
							isect.t = num_den;
							isect.temp0 = s;
							isect.temp1 = t;

							isect.ray.origin = ray.origin + ray.direction * isect.t;

							isect.normal = (objects[k].n0 + (objects[k].n1 - objects[k].n0)*s + (objects[k].n2 - objects[k].n0)*t).unit();
						}

					}

					if (isect.intersects && isect.t < se.tmax) {
						if (!intersect.intersects || isect.t < intersect.t) {
							intersect = isect;
							which = k;
						}
					}
				}
			}
		}


		
		
		for (int j = 0; j < object_count; j++) {

			int k = j;

			isect.intersects = false;

			if (objects[k].type == PLANE) {

				double den = ray.direction * objects[k].normal;
				if (fabs(den) < epsilon) continue;

				double num = (objects[k].point - ray.origin) * objects[k].normal;
				double num_den = num/den;
				if (num_den < epsilon) continue;

				isect.intersects = true;
				isect.t = num_den;

				isect.ray.origin = ray.origin + ray.direction * isect.t;

				isect.normal = objects[k].normal.unit();
				
			} else break;

			if (isect.intersects) {
				if (!intersect.intersects || isect.t < intersect.t) {
					intersect = isect;
					which = k;
				}
			}
		}


		

		if (intersect.intersects) {

			__a[l] = objects[which].emission;
			__b[l] = objects[which].color;

			// texture mapping
			if (objects[which].type == PLANE && objects[which].texture > -1) {
				__vector v0 = objects[which].e0;
				__vector v1 = objects[which].e1;
				__vector p  = (intersect.ray.origin - objects[which].point);

				double pv0 = p*v0;
				double pv1 = p*v1;
				double v0v0 = v0*v0;
				double v0v1 = v0*v1;
				double v1v1 = v1*v1;
				double den = v0v0*v1v1 - v0v1*v0v1;
				double s = (pv0*v1v1 - pv1*v0v1)/den;
				double t = (pv1*v0v0 - pv0*v0v1)/den;

				s = fabs(s);
				t = fabs(t);
				s = s - int(s);
				t = t - int(t);
				int twidth  = texture.width[objects[which].texture];
				int theight = texture.height[objects[which].texture];
				int tb      = texture.bpp[objects[which].texture] / 8;
				s *= twidth - 1;
				t *= theight - 1;
				unsigned int  _t =  t,      _s =  s,
				             __t = _t + 1, __s = _s + 1;
				 _s =  _s > (twidth)  - 2 ? (twidth  - 2) :  _s;
				__s = __s > (twidth)  - 1 ? (twidth  - 1) : __s;
				 _t =  _t > (theight) - 2 ? (theight - 2) :  _t;
				__t = __t > (theight) - 1 ? (theight - 1) : __t;

				__vector _a = __vector(texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 2]/256.0) * (__t -  t) * (__s -  s);
				__vector _b = __vector(texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 1]/256.0, 
						       texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 2]/256.0) * (  t - _t) * (__s -  s);
				__vector _c = __vector(texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 2]/256.0) * (__t -  t) * (  s - _s);
				__vector _d = __vector(texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 2]/256.0) * (  t - _t) * (  s - _s);

				__a[l] = __a[l].h(_a + _b + _c + _d);
				__b[l] = __b[l].h(_a + _b + _c + _d);
			} else if (objects[which].type == SPHERE && objects[which].texture > -1) {
				__vector v0 = objects[which].e0.unit();
				__vector v1 = objects[which].e1.unit();
				__vector p  = (intersect.ray.origin - objects[which].center).unit();

				// rework this
				double phi = acos(v0*p);
				double t = phi/3.14159265359;
				double theta = acos(v1*p/sin(phi))/(2*3.14159265359);
				double s = v0.cross(v1)*p > 0.0 ? theta : (1.0-theta);

				int twidth  = texture.width[objects[which].texture];
				int theight = texture.height[objects[which].texture];
				int tb      = texture.bpp[objects[which].texture] / 8;
				s *= twidth - 1;
				t *= theight - 1;
				unsigned int  _t =  t,      _s =  s,
				             __t = _t + 1, __s = _s + 1;
				 _s =  _s > (twidth)  - 2 ? (twidth  - 2) :  _s;
				__s = __s > (twidth)  - 1 ? (twidth  - 1) : __s;
				 _t =  _t > (theight) - 2 ? (theight - 2) :  _t;
				__t = __t > (theight) - 1 ? (theight - 1) : __t;

				__vector _a = __vector(texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 2]/256.0) * (__t -  t) * (__s -  s);
				__vector _b = __vector(texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 1]/256.0, 
						       texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 2]/256.0) * (  t - _t) * (__s -  s);
				__vector _c = __vector(texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 2]/256.0) * (__t -  t) * (  s - _s);
				__vector _d = __vector(texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 2]/256.0) * (  t - _t) * (  s - _s);

				__a[l] = __a[l].h(_a + _b + _c + _d);
				__b[l] = __b[l].h(_a + _b + _c + _d);
			} else if (objects[which].type == TRIANGLE && objects[which].texture > -1) {
				double s = intersect.temp0;
				double t = intersect.temp1;

				__vector v0 = objects[which].e1 - objects[which].e0;
				__vector v1 = objects[which].e2 - objects[which].e0;

				__vector texcoord = objects[which].e0 + v0 * s + v1 * t;
				s = texcoord.x;
				t = 1.0 - texcoord.y;
				
				int twidth  = texture.width[objects[which].texture];
				int theight = texture.height[objects[which].texture];
				int tb      = texture.bpp[objects[which].texture] / 8;
				s *= twidth - 1;
				t *= theight - 1;
				unsigned int  _t =  t,      _s =  s,
				             __t = _t + 1, __s = _s + 1;
				 _s =  _s > (twidth)  - 2 ? (twidth  - 2) :  _s;
				__s = __s > (twidth)  - 1 ? (twidth  - 1) : __s;
				 _t =  _t > (theight) - 2 ? (theight - 2) :  _t;
				__t = __t > (theight) - 1 ? (theight - 1) : __t;

				__vector _a = __vector(texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb +  _s*tb + 2]/256.0) * (__t -  t) * (__s -  s);
				__vector _b = __vector(texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 1]/256.0, 
						       texture.texture_device[objects[which].texture][__t*twidth*tb +  _s*tb + 2]/256.0) * (  t - _t) * (__s -  s);
				__vector _c = __vector(texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][ _t*twidth*tb + __s*tb + 2]/256.0) * (__t -  t) * (  s - _s);
				__vector _d = __vector(texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 0]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 1]/256.0,
						       texture.texture_device[objects[which].texture][__t*twidth*tb + __s*tb + 2]/256.0) * (  t - _t) * (  s - _s);

				__a[l] = __a[l].h(_a + _b + _c + _d);
				__b[l] = __b[l].h(_a + _b + _c + _d);
			}


			if (objects[which].material == DIFFUSE) {

				__vector w = intersect.normal;

				// cosine weighted sampling
				double u1 = rand_device[rand_size*(l+1) + rand_index + 0];
				double u2 = rand_device[rand_size*(l+1) + rand_index + 1];
				double r1 = 2 * M_PI * u1;
				double r2 = sqrt(1 - u2);
				double r3 = sqrt(u2);

				__vector u(0,0,0);
				if      (fabs(w.x) < fabs(w.y) + epsilon && fabs(w.x) < fabs(w.z) + epsilon) u.x = 1;
				else if (fabs(w.y) < fabs(w.x) + epsilon && fabs(w.y) < fabs(w.z) + epsilon) u.y = 1;
				else u.z = 1;

				u = u.cross(w).unit();
				__vector v = w.cross(u).unit();
					 u = v.cross(w).unit();
				__vector d = ( u * (cos(r1) * r2) + v * (sin(r1) * r2) + w * r3 ) .unit();

				intersect.ray.direction = d;
				ray = intersect.ray;

			} else if (objects[which].material == SPECULAR) {

				intersect.ray.direction = (ray.direction - intersect.normal * (ray.direction * intersect.normal * 2.0)).unit();
				ray = intersect.ray;
				
			} else if (objects[which].material == REFRACTIVE) {

				bool into = ray.direction * intersect.normal < 0; // entering the medium

				double n1 = into ? 1.0 : 1.5;
				double n2 = into ? 1.5 : 1.0;
				double n1n2 = n1/n2;
				__vector n  = into ? intersect.normal : (intersect.normal * -1);
				__vector r = ray.direction;
				
				double n1n22 = n1n2 * n1n2;
				double rn   = r * n;
				double rn2  = rn * rn;
				
				double a = 1 - n1n22 * (1 - rn2);
				if (a >= 0) {
					// fresnel
					double a_sqrt = sqrt(a);
					double Rs = ((n1 * rn * -1) - (n2 * a_sqrt)) / ((n1 * rn * -1) + (n2 * a_sqrt));
					Rs *= Rs;
					double Rp = ((n1 * a_sqrt) - (n2 * rn * -1)) / ((n1 * a_sqrt) + (n2 * rn * -1));
					Rp *= Rp;
					double R = (Rs + Rp)/2.0;
					//double T = 1 - R;
					if (rand_device[rand_size*(l+1) + rand_index + 2] < R) {	// reflect
						intersect.ray.direction = (ray.direction - intersect.normal * (ray.direction * intersect.normal * 2.0)).unit();
						ray = intersect.ray;
					} else {							// transmit
						//ray.origin = intersect.ray.origin;
						//ray.direction = r * n1n2 - n * (n1n2 * rn + a_sqrt);
						intersect.ray.direction = r * n1n2 - n * (n1n2 * rn + a_sqrt);
						ray = intersect.ray;
					}
				} else {
					  intersect.ray.direction = (ray.direction - intersect.normal * (ray.direction * intersect.normal * 2.0)).unit();
					  ray = intersect.ray; // total internal reflection
				}
			}
			
		} else break;
	}

	sample = __a[max_bounces - 1];
	for (int l = max_bounces-2; l >= 0; l--) sample = __a[l] + __b[l].h(sample);

	return sample;
}

__global__ void kernel(__node n, __buffers b, int samples, int max_bounces, __dimensions dim, __offsets offset, __camera cmodel, __textures texture) {
  	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int rand_index = y * 8 * 16 * 3 + x * 3;

	int index = ((y + offset.y) * dim.width + (dim.width - x - offset.x - 1)) * 3;
	if (index < 0 || index >= dim.height * dim.width * 3) return;

	double u1 = b.rand_device[rand_index + 0];
	double u2 = b.rand_device[rand_index + 1];
	double r1 = 2 * M_PI * u1;
	double r2 = sqrt(1 - u2);
	__vector o = __vector( (x + offset.x) - dim.width  / 2,
			       (y + offset.y) - dim.height / 2,
			                                    0) + __vector(cos(r1)*r2, sin(r1)*r2, 0) * 0.5;
	
	o.x =   o.x/dim.width  * 36.0/1000.0;			// 36mm sensor
	o.y =   o.y/dim.height * 36.0/1000.0 * dim.height/dim.width;
	o.z =   cmodel.image_distance;

	__ray ray;
	ray.origin = o;
	ray.direction = o.unit() * -1;

	__vector p = ray.direction * (cmodel.focal_distance / fabs(ray.direction.z));

	u1 = b.rand_device[rand_index+1];
	u2 = b.rand_device[rand_index+2];
	r1 = 2 * M_PI * u1;
	r2 = u2;
	
	ray.origin = __vector(cos(r1)*r2, sin(r1)*r2, 0) * cmodel.aperture_diameter * 0.5;
	ray.direction = (p - ray.origin).unit();

	// sample the ray
	//__vector _sample = sampleRay(b, ray, objects, object_count, 0, rand_device, rand_index, dim.width*dim.height*3, max_bounces, texture);
	__vector _sample = sampleRay(n, ray, b.objects_device, b.object_count, 0, b.rand_device, rand_index, 16 * 8 * 16 * 8 * 3, max_bounces, texture);

	// add the sample to the  accumulation
	b.doubles_device[index+0] = (b.doubles_device[index + 0] * (samples - 1.0) + _sample.x) / samples;
	b.doubles_device[index+1] = (b.doubles_device[index + 1] * (samples - 1.0) + _sample.y) / samples;
	b.doubles_device[index+2] = (b.doubles_device[index + 2] * (samples - 1.0) + _sample.z) / samples;

	// save the current frame
	b.char_device[index+0] = CLAMP((int)(b.doubles_device[index + 0] * 255.0), 0, 255);
	b.char_device[index+1] = CLAMP((int)(b.doubles_device[index + 1] * 255.0), 0, 255);
	b.char_device[index+2] = CLAMP((int)(b.doubles_device[index + 2] * 255.0), 0, 255);
}

bool initializePathTracer(__node& n, __buffers& b, int max_bounces, __dimensions dim, __bounds bounds, __textures& texture) {

	int num_bytes;

// allocate tree -------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.id_device, (sizeof(int) * n.size))   ));
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.leaf_id_device, (sizeof(int) * n.size))   ));
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.parent_device, (sizeof(int) * n.size))   ));
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.child0_device, (sizeof(int) * n.size))   ));
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.child1_device, (sizeof(int) * n.size))   ));
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.bounds_device, (sizeof(__bounds) * n.size))   ));
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.axis_device, (sizeof(char) * n.size))   ));
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.object_count_device, (sizeof(int) * n.size))   ));
	printf("%s\n", hipGetErrorString(  hipMalloc((void**)&n.objects_device, (sizeof(int) * n.leaf_size * n.max_leaf_objects))   ));
// --------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

	// for storing the image result
	num_bytes = sizeof(unsigned char) * dim.width * dim.height * 3;
	b.char_host = (unsigned char*)malloc(num_bytes);
	hipMalloc((void**)&(b.char_device), num_bytes);

	// for accumulating samples
	num_bytes = sizeof(double) * dim.width * dim.height * 3;
	b.doubles_host = (double*)malloc(num_bytes);
	hipMalloc((void**)&(b.doubles_device), num_bytes);

	// for storing the objects on the device
	num_bytes = sizeof(__object) * b.object_count;
	hipMalloc((void**)&b.objects_device, num_bytes);

	// for generating random uniforms
	int m = 1 + max_bounces;
//	num_bytes = sizeof(float) * dim.width * dim.height * 3 * m;
	num_bytes = sizeof(float) * 16 * 8 * 16 * 8 * 3 * m;
	hipMalloc((void**)&b.rand_device, num_bytes);
	
	// for storing the textures
	for (int i = 0; i < texture.count; i++) {
		num_bytes = sizeof(unsigned char) * texture.width[i] * texture.height[i] * texture.bpp[i] / 8;
		hipMalloc((void**)&(texture.texture_device[i]), num_bytes);
		hipMemcpy(texture.texture_device[i], texture.texture[i], num_bytes, hipMemcpyHostToDevice);
	}

	return true;
}

bool setupPathTracer(__node& n, __buffers& b, __dimensions dim, __bounds bounds) {

	int num_bytes;

// copy tree -------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.id_device, n.id, sizeof(int) * n.size, hipMemcpyHostToDevice)  ));
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.leaf_id_device, n.leaf_id, sizeof(int) * n.size, hipMemcpyHostToDevice)  ));
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.parent_device, n.parent, sizeof(int) * n.size, hipMemcpyHostToDevice)  ));
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.child0_device, n.child0, sizeof(int) * n.size, hipMemcpyHostToDevice)  ));
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.child1_device, n.child1, sizeof(int) * n.size, hipMemcpyHostToDevice)  ));
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.bounds_device, n.bounds, sizeof(__bounds) * n.size, hipMemcpyHostToDevice)  ));
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.axis_device, n.axis, sizeof(char) * n.size, hipMemcpyHostToDevice)  ));
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.object_count_device, n.object_count, sizeof(int) * n.size, hipMemcpyHostToDevice)  ));
	printf("%s\n", hipGetErrorString(  hipMemcpy(n.objects_device, n.objects, sizeof(int) * n.leaf_size * n.max_leaf_objects, hipMemcpyHostToDevice)  ));
// ------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

	// clear the image result
	num_bytes = sizeof(unsigned char) * dim.width * dim.height * 3;
	memset(b.char_host, 0, num_bytes);
	hipMemset(b.char_device, 0, num_bytes);
	// zero out the sample accumulation
	num_bytes = sizeof(double) * dim.width * dim.height * 3;
	memset(b.doubles_host, 0, num_bytes);
	hipMemset(b.doubles_device, 0, num_bytes);
	// store the objects on the device
	num_bytes = sizeof(__object) * b.object_count;
	hipMemcpy(b.objects_device, b.objects_host, num_bytes, hipMemcpyHostToDevice);

	return true;
}

bool releasePathTracer(__node& n, __buffers& b, __textures& texture) {

///////////////////
	hipFree(n.id_device);
	hipFree(n.leaf_id_device);
	hipFree(n.parent_device);
	hipFree(n.child0_device);
	hipFree(n.child1_device);
	hipFree(n.bounds_device);
	hipFree(n.axis_device);
	hipFree(n.object_count_device);
	hipFree(n.objects_device);
///////////////////

	free(b.char_host);
	hipFree(b.char_device);
	free(b.doubles_host);
	hipFree(b.doubles_device);
	hipFree(b.objects_device);
	hipFree(b.rand_device);

	for (int i = 0; i < texture.count; i++) hipFree(texture.texture_device[i]);

	return true;
}

bool runPathTracer(__node& n, __buffers b, int max_bounces, __dimensions dim, __offsets offset, unsigned long long samples, __camera cmodel, __textures& texture) {

	int dimx = 16;
	int dimy = 16;
	dim3 dimGrid(8,8);
//	dim3 dimGrid(dim.width/dimx, dim.height/dimy);
	dim3 dimBlock(dimx, dimy);

	// generate random uniforms
	if (offset.x == 0 && offset.y == 0) {
		int m = 1 + max_bounces;//4 + max_bounces * 4; // 1 plus 10 depth
		hiprandGenerator_t gen;
		hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
		hiprandSetPseudoRandomGeneratorSeed(gen, samples); //samples=seed
//		hiprandGenerateUniform(gen, rand_device, dim.width * dim.height * 3 * m);
		hiprandGenerateUniform(gen, b.rand_device, 16 * 8 * 16 * 8 * 3 * m);
		hiprandDestroyGenerator(gen);
	}
	
	//hipDeviceSetLimit (hipLimitStackSize, );
	//cudaThreadSetLimit (hipLimitStackSize, 8192*16)

	kernel<<<dimGrid, dimBlock>>>(n, b, samples, max_bounces, dim, offset, cmodel, texture);

	return true;
}

bool grabFrame(__buffers b, __dimensions dim) {
	int num_bytes;

	num_bytes = sizeof(unsigned char) * dim.width * dim.height * 3;
	printf("%s\n", hipGetErrorString( hipMemcpy(b.char_host, b.char_device, num_bytes, hipMemcpyDeviceToHost) ));

//	num_bytes = sizeof(double) * dim.width * dim.height * 3;
//	printf("%s\n", hipGetErrorString( hipMemcpy(b.doubles_host, b.doubles_device, num_bytes, hipMemcpyDeviceToHost) ));

	return true;
}